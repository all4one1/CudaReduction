#include "hip/hip_runtime.h"
#include "CuReduction.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include "nvidia_kernels.h"
__global__ void init_test(double* data, unsigned int n)
{
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < n)
	{
		data[i] = i + 1;
	}
}
__global__ void gpu_print(double* f)
{
	printf("message: %f", f[0]);
	printf("\n");
}
__global__ void reduction_abs_sum(double* data, unsigned int n, double* reduced) {
	extern __shared__ double shared[];


	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) {
		shared[tid] = abs(data[i]);
	}
	else
	{
		shared[tid] = 0.0;
	}

	__syncthreads();


	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			shared[tid] += shared[tid + s];
		}

		__syncthreads();
	}


	if (tid == 0) {
		reduced[blockIdx.x] = shared[0];
	}


}
__global__ void reduction_signed_sum(double* data, unsigned int n, double* reduced) {
	extern __shared__ double shared[];


	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) {
		shared[tid] = data[i];
	}
	else
	{
		shared[tid] = 0.0;
	}

	__syncthreads();


	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			shared[tid] += shared[tid + s];
		}

		__syncthreads();
	}


	if (tid == 0) {
		reduced[blockIdx.x] = shared[0];
	}
}

__global__ void dot_product(double* v1, double *v2, unsigned int n, double* reduced) {
	extern __shared__ double shared[];


	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) 
	{
		shared[tid] = v1[i] * v2[i];
	}
	else
	{
		shared[tid] = 0.0;
	}

	__syncthreads();


	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			shared[tid] += shared[tid + s];
		}

		__syncthreads();
	}


	if (tid == 0) {
		reduced[blockIdx.x] = shared[0];
	}
}

	


CudaReduction::CudaReduction()
{

}

CudaReduction::CudaReduction(unsigned int N, unsigned int thr)
{
	set_reduced_size(N, thr, true);

	if (res_array != nullptr) hipFree(res_array);
	hipMalloc((void**)&res_array, sizeof(double) * N_v[1]);

	if (arr != nullptr) delete[] arr;
	arr = new double* [steps + 1];
}

CudaReduction::CudaReduction(double* device_ptr, unsigned int N, unsigned int thr)
{
	set_reduced_size(N, thr, true);

	if (res_array != nullptr) hipFree(res_array);
	hipMalloc((void**)&res_array, sizeof(double) * N_v[1]);

	if (arr != nullptr) delete[] arr;
	arr = new double* [steps + 1];

	arr[0] = device_ptr;
	for (unsigned int i = 1; i <= steps; i++)
		arr[i] = res_array;
}


CudaReduction::~CudaReduction()
{
	hipFree(res_array); res_array = nullptr;
	delete[] arr; arr = nullptr;
	grid_v.clear();
	N_v.clear();
}

void CudaReduction::set_reduced_size(unsigned int N, unsigned int thr, bool doubleRead)
{
	if (thr < 64)
	{
		std::cout << "more threads needed " << std::endl;
		threads = 64;
	}

	unsigned int temp_ = N;
	threads = thr;
	N_v.push_back(N);

	steps = 0;
	while (true)
	{
		steps++;
		if (doubleRead) temp_ = (temp_ + (threads * 2 - 1)) / (threads * 2);
		else temp_ = (temp_ + threads - 1) / threads;

		grid_v.push_back(temp_);
		N_v.push_back(temp_);
		if (temp_ == 1)  break;
	}
}

void CudaReduction::print_check()
{
	gpu_print << <1, 1 >> > (res_array);
}

void CudaReduction::auto_test()
{
	double* ptr_d;
	int N = 123456;

	hipMalloc((void**)&ptr_d, N * sizeof(double));
	init_test << <1024, 1024 >> > (ptr_d, N);

	std::cout << std::fixed;
	std::cout << "Exact value = " << N / 2.0 * (N + 1)  << std::endl;
	std::cout << "Cuda result = " << CudaReduction::reduce(ptr_d, N, 128) << std::endl;
}


double CudaReduction::check_on_cpu(double* device_ptr, unsigned int N)
{
	double* f = new double[N];
	hipMemcpy(f, device_ptr, sizeof(double) * N, hipMemcpyDeviceToHost);
	double s = 0;
	for (unsigned int i = 0; i < N; i++)
		s += abs(f[i]);
	return s;
}


double CudaReduction::reduce_legacy(bool withCopy)
{
	switch (type)
	{
	case CudaReduction::ABSSUM:	
		for (unsigned int i = 0; i < steps; i++)
			reduction_abs_sum << < grid_v[i], threads, 1024 * sizeof(double) >> > (arr[i], N_v[i], arr[i + 1]);
		break;
	case CudaReduction::SIGNEDSUM:
		for (unsigned int i = 0; i < steps; i++)
			reduction_signed_sum << < grid_v[i], threads, 1024 * sizeof(double) >> > (arr[i], N_v[i], arr[i + 1]);
		break;
	case CudaReduction::DOTPRODUCT:
		// todo
		//for (unsigned int i = 0; i < steps; i++)
		//	dot_product << < Gp[i], threads, 1024 * sizeof(double) >> > (arr[i], arr[i], Np[i], arr[i + 1]); 
		break;
	default:
		break;
	}

	if (withCopy) hipMemcpy(&res, res_array, sizeof(double), hipMemcpyDeviceToHost);

	return res;
}

double CudaReduction::reduce(double* device_ptr, bool withCopy)
{
	arr[0] = device_ptr;
	for (unsigned int i = 1; i <= steps; i++)
		arr[i] = res_array;

	return reduce(withCopy);
}

double CudaReduction::reduce(double* device_ptr, unsigned int N, unsigned int thr, bool withCopy)
{
	CudaReduction temp(device_ptr, N, thr);
	return temp.reduce(withCopy);
}


double CudaReduction::reduce(bool withCopy)
{
	for (unsigned int i = 0; i < steps; i++)			
		reduce_<double>(N_v[i], threads, grid_v[i], 5, arr[i], arr[i + 1]);

	//for (unsigned int i = 0; i < steps; i++)
	//{
	//	void* args[] = { &arr[i], &arr[i + 1], &N_v[i] };
	//	hipLaunchKernel(reinterpret_cast<const void*>(kernel_ptr), grid_v[i], threads, args, threads * sizeof(double), 0);
	//}

	if (withCopy) hipMemcpy(&res, res_array, sizeof(double), hipMemcpyDeviceToHost);

	return res;
}


CuGraph CudaReduction::make_graph(double* device_ptr, bool withCopy)
{
	arr[0] = device_ptr;
	for (unsigned int i = 1; i <= steps; i++)
		arr[i] = res_array;

	CuGraph graph;
	for (unsigned int i = 0; i < steps; i++)
	{
		void* args[3] = { &arr[i], &N_v[i], &arr[i + 1] };
		graph.add_kernel_node(threads, grid_v[i], reduction_abs_sum, args, smem);
	}

	if (withCopy) graph.add_copy_node(&res, res_array, sizeof(double), hipMemcpyDeviceToHost);

	return graph;
}